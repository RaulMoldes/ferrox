#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <float.h>
#include "globals.cuh"

// ========== WARP REDUCTION PRIMITIVES =======
template <typename T>
__inline__ __device__ T warp_reduce_sum(T val) {
    // Use __shfl_down_sync for better performance than __shfl_xor_sync
#pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        val += __shfl_down_sync(0xffffffff, val, offset, 32);
    }
    return val;
}

template <typename T>
__inline__ __device__ T warp_reduce_max(T val) {
#pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        val = max(val, __shfl_down_sync(0xffffffff, val, offset, 32));
    }
    return val;
}

template <typename T>
__inline__ __device__ T warp_reduce_min(T val) {
#pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        val = min(val, __shfl_down_sync(0xffffffff, val, offset, 32));
    }
    return val;
}

template <typename T>
__inline__ __device__ T warp_reduce_prod(T val) {
#pragma unroll
    for (int offset = 16; offset > 0; offset >>= 1) {
        val *= __shfl_down_sync(0xffffffff, val, offset, 32);
    }
    return val;
}


// ========== KERNEL IMPLEMENTATIONS ==========

// Sum kernels
extern "C" __global__ void reduce_sum_all(const float* input, float* output, int size) {
    __shared__ float shared_data[BLOCK_SIZE];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = blockDim.x * gridDim.x;

    // Each thread processes multiple elements
    float thread_result = 0.0f;
    while (idx < size) {
        thread_result += input[idx];
        idx += grid_size;
    }

    shared_data[tid] = thread_result;
    __syncthreads();

    // Tree reduction in shared memory
    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}

extern "C" __global__ void reduce_sum_all_f64(const double* input, double* output, int size) {
    __shared__ double shared_data[BLOCK_SIZE];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = blockDim.x * gridDim.x;

    double thread_result = 0.0;
    while (idx < size) {
        thread_result += input[idx];
        idx += grid_size;
    }

    shared_data[tid] = thread_result;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] += shared_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}



extern "C" __global__ void reduce_sum_axes(
    const float* __restrict__ input,
    float* __restrict__ output,
    int outer_size,
    int axis_size,
    int inner_size
) {
    int output_idx = blockIdx.x;

    if (output_idx >= outer_size * inner_size) return;

    int outer_idx = output_idx / inner_size;
    int inner_idx = output_idx % inner_size;

    float sum = 0.0f;

    // Each thread processes multiple elements along axis to improve memory efficiency
    for (int axis_idx = threadIdx.x; axis_idx < axis_size; axis_idx += blockDim.x) {
        int input_idx = outer_idx * axis_size * inner_size + axis_idx * inner_size + inner_idx;
        sum += input[input_idx];
    }

    // Warp-level reduction
    sum = warp_reduce_sum(sum);

    // Block-level reduction using shared memory
    __shared__ float block_sums[32];
    int warp_id = threadIdx.x >> 5;
    int lane = threadIdx.x & 31;

    if (lane == 0) {
        block_sums[warp_id] = sum;
    }
    __syncthreads();

    // Final reduction
    if (warp_id == 0) {
        sum = (threadIdx.x < (blockDim.x >> 5)) ? block_sums[lane] : 0.0f;
        sum = warp_reduce_sum(sum);

        if (threadIdx.x == 0) {
            output[output_idx] = sum;
        }
    }
}

extern "C" __global__ void reduce_sum_axes_f64(
    const double* __restrict__ input,
    double* __restrict__ output,
    int outer_size,
    int axis_size,
    int inner_size
) {
    int output_idx = blockIdx.x;

    if (output_idx >= outer_size * inner_size) return;

    int outer_idx = output_idx / inner_size;
    int inner_idx = output_idx % inner_size;

    double sum = 0.0;

    for (int axis_idx = threadIdx.x; axis_idx < axis_size; axis_idx += blockDim.x) {
        int input_idx = outer_idx * axis_size * inner_size + axis_idx * inner_size + inner_idx;
        sum += input[input_idx];
    }

    sum = warp_reduce_sum(sum);

    __shared__ double block_sums[32];
    int warp_id = threadIdx.x >> 5;
    int lane = threadIdx.x & 31;

    if (lane == 0) {
        block_sums[warp_id] = sum;
    }
    __syncthreads();

    if (warp_id == 0) {
        sum = (threadIdx.x < (blockDim.x >> 5)) ? block_sums[lane] : 0.0;
        sum = warp_reduce_sum(sum);

        if (threadIdx.x == 0) {
            output[output_idx] = sum;
        }
    }
}


// Max kernels
extern "C" __global__ void reduce_max_all(const float* input, float* output, int size) {
    __shared__ float shared_data[BLOCK_SIZE];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = blockDim.x * gridDim.x;

    float thread_result = -INFINITY;
    while (idx < size) {
        thread_result = fmaxf(thread_result, input[idx]);
        idx += grid_size;
    }

    shared_data[tid] = thread_result;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] = fmaxf(shared_data[tid], shared_data[tid + stride]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}

extern "C" __global__ void reduce_max_all_f64(const double* input, double* output, int size) {
    __shared__ double shared_data[BLOCK_SIZE];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = blockDim.x * gridDim.x;

    double thread_result = -INFINITY;
    while (idx < size) {
        thread_result = fmax(thread_result, input[idx]);
        idx += grid_size;
    }

    shared_data[tid] = thread_result;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] = fmax(shared_data[tid], shared_data[tid + stride]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}


extern "C" __global__ void reduce_max_axes(
    const float* __restrict__ input,
    float* __restrict__ output,
    int outer_size,
    int axis_size,
    int inner_size
) {
    int output_idx = blockIdx.x;

    if (output_idx >= outer_size * inner_size) return;

    int outer_idx = output_idx / inner_size;
    int inner_idx = output_idx % inner_size;

    float max_val = -FLT_MAX;

    for (int axis_idx = threadIdx.x; axis_idx < axis_size; axis_idx += blockDim.x) {
        int input_idx = outer_idx * axis_size * inner_size + axis_idx * inner_size + inner_idx;
        max_val = max(max_val, input[input_idx]);
    }

    max_val = warp_reduce_max(max_val);

    __shared__ float block_maxs[32];
    int warp_id = threadIdx.x >> 5;
    int lane = threadIdx.x & 31;

    if (lane == 0) {
        block_maxs[warp_id] = max_val;
    }
    __syncthreads();

    if (warp_id == 0) {
        max_val = (threadIdx.x < (blockDim.x >> 5)) ? block_maxs[lane] : -FLT_MAX;
        max_val = warp_reduce_max(max_val);

        if (threadIdx.x == 0) {
            output[output_idx] = max_val;
        }
    }
}

extern "C" __global__ void reduce_max_axes_f64(
    const double* __restrict__ input,
    double* __restrict__ output,
    int outer_size,
    int axis_size,
    int inner_size
) {
    int output_idx = blockIdx.x;

    if (output_idx >= outer_size * inner_size) return;

    int outer_idx = output_idx / inner_size;
    int inner_idx = output_idx % inner_size;

    double max_val = -DBL_MAX;

    for (int axis_idx = threadIdx.x; axis_idx < axis_size; axis_idx += blockDim.x) {
        int input_idx = outer_idx * axis_size * inner_size + axis_idx * inner_size + inner_idx;
        max_val = max(max_val, input[input_idx]);
    }

    max_val = warp_reduce_max(max_val);

    __shared__ double block_maxs[32];
    int warp_id = threadIdx.x >> 5;
    int lane = threadIdx.x & 31;

    if (lane == 0) {
        block_maxs[warp_id] = max_val;
    }
    __syncthreads();

    if (warp_id == 0) {
        max_val = (threadIdx.x < (blockDim.x >> 5)) ? block_maxs[lane] : -DBL_MAX;
        max_val = warp_reduce_max(max_val);

        if (threadIdx.x == 0) {
            output[output_idx] = max_val;
        }
    }
}



// Min kernels
extern "C" __global__ void reduce_min_all(const float* input, float* output, int size) {
    __shared__ float shared_data[BLOCK_SIZE];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = blockDim.x * gridDim.x;

    float thread_result = INFINITY;
    while (idx < size) {
        thread_result = fminf(thread_result, input[idx]);
        idx += grid_size;
    }

    shared_data[tid] = thread_result;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] = fminf(shared_data[tid], shared_data[tid + stride]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}

extern "C" __global__ void reduce_min_all_f64(const double* input, double* output, int size) {
    __shared__ double shared_data[BLOCK_SIZE];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = blockDim.x * gridDim.x;

    double thread_result = INFINITY;
    while (idx < size) {
        thread_result = fmin(thread_result, input[idx]);
        idx += grid_size;
    }

    shared_data[tid] = thread_result;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] = fmin(shared_data[tid], shared_data[tid + stride]);
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}


extern "C" __global__ void reduce_min_axes(
    const float* __restrict__ input,
    float* __restrict__ output,
    int outer_size,
    int axis_size,
    int inner_size
) {
    int output_idx = blockIdx.x;

    if (output_idx >= outer_size * inner_size) return;

    int outer_idx = output_idx / inner_size;
    int inner_idx = output_idx % inner_size;

    float min_val = FLT_MAX;

    for (int axis_idx = threadIdx.x; axis_idx < axis_size; axis_idx += blockDim.x) {
        int input_idx = outer_idx * axis_size * inner_size + axis_idx * inner_size + inner_idx;
        min_val = min(min_val, input[input_idx]);
    }

    min_val = warp_reduce_min(min_val);

    __shared__ float block_mins[32];
    int warp_id = threadIdx.x >> 5;
    int lane = threadIdx.x & 31;

    if (lane == 0) {
        block_mins[warp_id] = min_val;
    }
    __syncthreads();

    if (warp_id == 0) {
        min_val = (threadIdx.x < (blockDim.x >> 5)) ? block_mins[lane] : FLT_MAX;
        min_val = warp_reduce_min(min_val);

        if (threadIdx.x == 0) {
            output[output_idx] = min_val;
        }
    }
}

extern "C" __global__ void reduce_min_axes_f64(
    const double* __restrict__ input,
    double* __restrict__ output,
    int outer_size,
    int axis_size,
    int inner_size
) {
    int output_idx = blockIdx.x;

    if (output_idx >= outer_size * inner_size) return;

    int outer_idx = output_idx / inner_size;
    int inner_idx = output_idx % inner_size;

    double min_val = DBL_MAX;

    for (int axis_idx = threadIdx.x; axis_idx < axis_size; axis_idx += blockDim.x) {
        int input_idx = outer_idx * axis_size * inner_size + axis_idx * inner_size + inner_idx;
        min_val = min(min_val, input[input_idx]);
    }

    min_val = warp_reduce_min(min_val);

    __shared__ double block_mins[32];
    int warp_id = threadIdx.x >> 5;
    int lane = threadIdx.x & 31;

    if (lane == 0) {
        block_mins[warp_id] = min_val;
    }
    __syncthreads();

    if (warp_id == 0) {
        min_val = (threadIdx.x < (blockDim.x >> 5)) ? block_mins[lane] : DBL_MAX;
        min_val = warp_reduce_min(min_val);

        if (threadIdx.x == 0) {
            output[output_idx] = min_val;
        }
    }
}

// Product kernels
extern "C" __global__ void reduce_prod_all(const float* input, float* output, int size) {
    __shared__ float shared_data[BLOCK_SIZE];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = blockDim.x * gridDim.x;

    float thread_result = 1.0f;
    while (idx < size) {
        thread_result *= input[idx];
        idx += grid_size;
    }

    shared_data[tid] = thread_result;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] *= shared_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}

extern "C" __global__ void reduce_prod_all_f64(const double* input, double* output, int size) {
    __shared__ double shared_data[BLOCK_SIZE];

    int tid = threadIdx.x;
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int grid_size = blockDim.x * gridDim.x;

    double thread_result = 1.0;
    while (idx < size) {
        thread_result *= input[idx];
        idx += grid_size;
    }

    shared_data[tid] = thread_result;
    __syncthreads();

    for (int stride = blockDim.x / 2; stride > 0; stride >>= 1) {
        if (tid < stride) {
            shared_data[tid] *= shared_data[tid + stride];
        }
        __syncthreads();
    }

    if (tid == 0) {
        output[blockIdx.x] = shared_data[0];
    }
}



extern "C" __global__ void reduce_prod_axes(
    const float* __restrict__ input,
    float* __restrict__ output,
    int outer_size,
    int axis_size,
    int inner_size
) {
    int output_idx = blockIdx.x;

    if (output_idx >= outer_size * inner_size) return;

    int outer_idx = output_idx / inner_size;
    int inner_idx = output_idx % inner_size;

    float prod = 1.0f;

    for (int axis_idx = threadIdx.x; axis_idx < axis_size; axis_idx += blockDim.x) {
        int input_idx = outer_idx * axis_size * inner_size + axis_idx * inner_size + inner_idx;
        prod *= input[input_idx];
    }

    prod = warp_reduce_prod(prod);

    __shared__ float block_prods[32];
    int warp_id = threadIdx.x >> 5;
    int lane = threadIdx.x & 31;

    if (lane == 0) {
        block_prods[warp_id] = prod;
    }
    __syncthreads();

    if (warp_id == 0) {
        prod = (threadIdx.x < (blockDim.x >> 5)) ? block_prods[lane] : 1.0f;
        prod = warp_reduce_prod(prod);

        if (threadIdx.x == 0) {
            output[output_idx] = prod;
        }
    }
}

extern "C" __global__ void reduce_prod_axes_f64(
    const double* __restrict__ input,
    double* __restrict__ output,
    int outer_size,
    int axis_size,
    int inner_size
) {
    int output_idx = blockIdx.x;

    if (output_idx >= outer_size * inner_size) return;

    int outer_idx = output_idx / inner_size;
    int inner_idx = output_idx % inner_size;

    double prod = 1.0;

    for (int axis_idx = threadIdx.x; axis_idx < axis_size; axis_idx += blockDim.x) {
        int input_idx = outer_idx * axis_size * inner_size + axis_idx * inner_size + inner_idx;
        prod *= input[input_idx];
    }

    prod = warp_reduce_prod(prod);

    __shared__ double block_prods[32];
    int warp_id = threadIdx.x >> 5;
    int lane = threadIdx.x & 31;

    if (lane == 0) {
        block_prods[warp_id] = prod;
    }
    __syncthreads();

    if (warp_id == 0) {
        prod = (threadIdx.x < (blockDim.x >> 5)) ? block_prods[lane] : 1.0;
        prod = warp_reduce_prod(prod);

        if (threadIdx.x == 0) {
            output[output_idx] = prod;
        }
    }
}
