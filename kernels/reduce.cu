
#include <hip/hip_runtime.h>
// kernels/max_along_dim.cu
__device__ inline int get_global_idx() {
    return blockIdx.x * blockDim.x + threadIdx.x;
}

// Simplified max reduction along a dimension
extern "C" __global__ void max_along_dim(
    const float* input,
    float* output,
    int outer_size,
    int axis_size,
    int inner_size
) {
    int idx = get_global_idx();
    int total_output = outer_size * inner_size;
    
    if (idx >= total_output) return;
    
    // Calculate position in output
    int outer_idx = idx / inner_size;
    int inner_idx = idx % inner_size;
    
    // Find maximum along the axis
    float max_val = -INFINITY;
    for (int axis_idx = 0; axis_idx < axis_size; axis_idx++) {
        int input_idx = outer_idx * axis_size * inner_size + axis_idx * inner_size + inner_idx;
        float val = input[input_idx];
        if (val > max_val) {
            max_val = val;
        }
    }
    
    output[idx] = max_val;
}

extern "C" __global__ void sum_axis(
    const float* input,
    float* output,
    int outer_size,
    int axis_size, 
    int inner_size
) {
    int outer_idx = blockIdx.x;
    int inner_idx = threadIdx.x;
    
    if (outer_idx >= outer_size || inner_idx >= inner_size) return;
    
    float sum = 0.0f;
    for (int axis_idx = 0; axis_idx < axis_size; axis_idx++) {
        int input_idx = outer_idx * axis_size * inner_size + 
                       axis_idx * inner_size + inner_idx;
        sum += input[input_idx];
    }
    
    int output_idx = outer_idx * inner_size + inner_idx;
    output[output_idx] = sum;
}
