
#include <hip/hip_runtime.h>
extern "C" __global__ void elementwise_mul(const float* a, const float* b, float* c, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        c[idx] = a[idx] * b[idx];
    }
}
