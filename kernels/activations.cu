
#include <hip/hip_runtime.h>
// Helper function to get global thread index
__device__ inline int get_global_idx() {
    return  get_global_idx();;
}

extern "C" __global__ void relu(
    const float* input,
    float* output,
    int size
) {
    int idx = get_global_idx();
    
    if (idx < size) {
        output[idx] = fmaxf(0.0f, input[idx]);
    }
}

extern "C" __global__ void sigmoid(const float* input, float* output, int size) {
    int idx = get_global_idx();
    if (idx < size) {
        // Element-wise sigmoid function: output = 1 / (1 + exp(-input))
        float x = input[idx];
        output[idx] = 1.0f / (1.0f + expf(-x));
    }
}


extern "C" __global__ void hyperbolic_tangent(
    const float* input,
    float* output,
    int size
) {
    // Calculate thread index using standard CUDA pattern
    int idx =  get_global_idx();
    
    // Boundary check for thread safety
    if (idx < size) {
        float x = input[idx];
        
        // Use the built-in tanhf function for numerical stability
        // It handles overflow/underflow cases internally
        output[idx] = tanhf(x);
        
        // Alternative manual implementation (commented for reference):
        // float exp_pos = expf(x);
        // float exp_neg = expf(-x);
        // output[idx] = (exp_pos - exp_neg) / (exp_pos + exp_neg);
    }
}