#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <float.h>
#include "globals.cuh"

extern "C" __global__ void relu(
    const float* input,
    float* output,
    int size
) {
    int idx = get_global_idx();

    if (idx < size) {
        output[idx] = fmaxf(0.0f, input[idx]);
    }
}



extern "C" __global__ void sigmoid(const float* input, float* output, int size) {
    int idx = get_global_idx();
    if (idx < size) {
        // Element-wise sigmoid function: output = 1 / (1 + exp(-input))
        float x = input[idx];
        output[idx] = 1.0f / (1.0f + expf(-x));
    }
}



extern "C" __global__ void sigmoid_f64(const double* input, double* output, int size) {
    int idx = get_global_idx();
    if (idx < size) {
        // Element-wise sigmoid function: output = 1 / (1 + exp(-input))
        double x = input[idx];
        output[idx] = 1.0f / (1.0f + expf(-x));
    }
}


extern "C" __global__ void hyperbolic_tangent(
    const float* input,
    float* output,
    int size
) {
    // Calculate thread index using standard CUDA pattern
    int idx = get_global_idx();

    // Boundary check for thread safety
    if (idx < size) {
        float x = input[idx];

        // Use the built-in tanhf function for numerical stability
        // It handles overflow/underflow cases internally
        output[idx] = tanhf(x);

        // Alternative manual implementation (commented for reference):
        // float exp_pos = expf(x);
        // float exp_neg = expf(-x);
        // output[idx] = (exp_pos - exp_neg) / (exp_pos + exp_neg);
    }
}

/// ADAPTED TO SUPPORT DOUBLE PRECISION
extern "C" __global__ void relu_f64(
    const double* input,
    double* output,
    int size
) {
    int idx = get_global_idx();
    if (idx < size) {
        output[idx] = fmax(0.0, input[idx]);  // 0.0 not 0.0d
    }
}

extern "C" __global__ void hyperbolic_tangent_f64(
    const double* input,
    double* output,
    int size
) {
    // Calculate thread index using standard CUDA pattern
    int idx = get_global_idx();

    // Boundary check for thread safety
    if (idx < size) {
        double x = input[idx];

        // Use the built-in tanhf function for numerical stability
        // It handles overflow/underflow cases internally
        output[idx] = tanh(x);

        // Alternative manual implementation (commented for reference):
        // float exp_pos = expf(x);
        // float exp_neg = expf(-x);
        // output[idx] = (exp_pos - exp_neg) / (exp_pos + exp_neg);
    }
}




// =============================================================================
// SOFTMAX KERNEL: WARP REDUCTION PRIMITIVES
// =============================================================================

template <typename T>
__inline__ __device__ T warpReduceMax(T val) {
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        val = max(val, __shfl_xor_sync(0xffffffff, val, mask, 32));
    }
    return val;
}

template <typename T>
__inline__ __device__ T warpReduceSum(T val) {
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        val += __shfl_xor_sync(0xffffffff, val, mask, 32);
    }
    return val;
}

// =============================================================================
// BLOCK REDUCTION PRIMITIVES
// =============================================================================

template <typename T>
__inline__ __device__ T blockReduceMax(T val) {
    __shared__ T shared[33];

    int lane = threadIdx.x & 0x1f;  // threadIdx.x % 32
    int wid = threadIdx.x >> 5;     // threadIdx.x / 32

    // Reduce inside the warp
    val = warpReduceMax(val);

    // Thread 0 of each warp stores on shmem
    if (lane == 0) {
        shared[wid] = val;
    }
    __syncthreads();

    // First warp makes the final reduction
    if (wid == 0) {
        val = (threadIdx.x < (blockDim.x / 32)) ? shared[lane] : -FLT_MAX;
        val = warpReduceMax(val);
    }

    return val;
}

template <typename T>
__inline__ __device__ T blockReduceSum(T val) {
    __shared__ T shared[33];

    int lane = threadIdx.x & 0x1f;
    int wid = threadIdx.x >> 5;

    val = warpReduceSum(val);

    if (lane == 0) {
        shared[wid] = val;
    }
    __syncthreads();

    if (wid == 0) {
        val = (threadIdx.x < (blockDim.x / 32)) ? shared[lane] : 0.0f;
        val = warpReduceSum(val);
    }

    return val;
}

// =============================================================================
// SOFTMAX KERNELS
// =============================================================================

template <typename T>
__device__ void softmax_kernel(const T* input, T* output, int N) {


    int tid = threadIdx.x;
    int global_idx = get_global_idx();

    // S1: Find global maxima using block reduction
    T local_max = -(T)FLT_MAX;

    // Initialize local max values.
    for (int i = global_idx; i < N; i += blockDim.x * gridDim.x) {
        local_max = max(local_max, input[i]);
    }

    // Reduce the maximum
    T block_max = blockReduceMax(local_max);

    // Broadcast the maximum to all threads
    __shared__ T s_max;
    if (tid == 0) s_max = block_max;
    __syncthreads();


    // S2: Compute local sums
    T local_sum = 0.0;

    // Each thread calculates its local sum value.
    for (int i = global_idx; i < N; i += blockDim.x * gridDim.x) {
        local_sum += expf(input[i] - s_max);
    }

    // Reduce sum between all blocks.
    T block_sum = blockReduceSum(local_sum);

    // Broadcast the sum to all threads.
    __shared__ T s_sum;
    if (tid == 0) s_sum = block_sum;
    __syncthreads();


    // S3: Compute final softmax and return the result.
    for (int i = global_idx; i < N; i += blockDim.x * gridDim.x) {
        output[i] = expf(input[i] - s_max) / s_sum;
    }
}




extern "C" __global__ void softmax_f64(const double* input, double* output, int N) {
    softmax_kernel<double>(input, output, N);

}

extern "C" __global__ void softmax(const float* input, float* output, int N) {
    softmax_kernel<float>(input, output, N);

}



// BATCH-AWARE SOFTMAX KERNELS
// =============================================================================
// This kernel processes multiple sequences in parallel, computing softmax
// along the specified axis while maintaining batch efficiency
// Each block processes one sequence from the batch
template <typename T>
__device__ void softmax_batch_axis(
    const T* input,
    T* output,
    int batch_size,
    int seq_length,
    int inner_size,
    int total_elements
) {
    int block_id = blockIdx.x;
    int tid = threadIdx.x;

    int inner_idx = block_id % inner_size;
    int batch_idx = block_id / inner_size;

    if (batch_idx >= batch_size) return;

    int sequence_start = batch_idx * seq_length * inner_size + inner_idx;

    // Find maximum
    T local_max = -(T)FLT_MAX;

    for (int i = tid; i < seq_length; i += blockDim.x) {
        int global_idx = sequence_start + i * inner_size;
        local_max = max(local_max, input[global_idx]);
    }

    T block_max = blockReduceMax(local_max);

    __shared__ T s_max;
    if (tid == 0) s_max = block_max;
    __syncthreads();


    // Compute sum
    T local_sum = 0.0;

    for (int i = tid; i < seq_length; i += blockDim.x) {
        int global_idx = sequence_start + i * inner_size;
        local_sum += exp(input[global_idx] - s_max);
    }

    T block_sum = blockReduceSum(local_sum);

    __shared__ T s_sum;
    if (tid == 0) s_sum = block_sum;
    __syncthreads();




    // Final softmax computation
    for (int i = tid; i < seq_length; i += blockDim.x) {
        int global_idx = sequence_start + i * inner_size;
        output[global_idx] = exp(input[global_idx] - s_max) / s_sum;
    }
}



extern "C" __global__ void softmax_batch_axis(
    const float* input,
    float* output,
    int batch_size,
    int seq_length,     // Size of the axis we're computing softmax over
    int inner_size,     // Size of dimensions after the softmax axis
    int total_elements
) {
    softmax_batch_axis<float>(input, output, batch_size, seq_length, inner_size, total_elements);
}


extern "C" __global__ void softmax_batch_axis_f64(
    const double* input,
    double* output,
    int batch_size,
    int seq_length,     // Size of the axis we're computing softmax over
    int inner_size,     // Size of dimensions after the softmax axis
    int total_elements
) {
    softmax_batch_axis<double>(input, output, batch_size, seq_length, inner_size, total_elements);
}
