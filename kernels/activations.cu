#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <math.h>
#include <float.h>
#include "globals.cuh"

extern "C" __global__ void relu(
    const float* input,
    float* output,
    int size
) {
    int idx = get_global_idx();

    if (idx < size) {
        output[idx] = fmaxf(0.0f, input[idx]);
    }
}

extern "C" __global__ void sigmoid(const float* input, float* output, int size) {
    int idx = get_global_idx();
    if (idx < size) {
        // Element-wise sigmoid function: output = 1 / (1 + exp(-input))
        float x = input[idx];
        output[idx] = 1.0f / (1.0f + expf(-x));
    }
}


extern "C" __global__ void hyperbolic_tangent(
    const float* input,
    float* output,
    int size
) {
    // Calculate thread index using standard CUDA pattern
    int idx =  get_global_idx();

    // Boundary check for thread safety
    if (idx < size) {
        float x = input[idx];

        // Use the built-in tanhf function for numerical stability
        // It handles overflow/underflow cases internally
        output[idx] = tanhf(x);

        // Alternative manual implementation (commented for reference):
        // float exp_pos = expf(x);
        // float exp_neg = expf(-x);
        // output[idx] = (exp_pos - exp_neg) / (exp_pos + exp_neg);
    }
}

/// ADAPTED TO SUPPORT DOUBLE PRECISION
extern "C" __global__ void relu_f64(
    const double* input,
    double* output,
    int size
) {
    int idx = get_global_idx();
    if (idx < size) {
        output[idx] = fmax(0.0, input[idx]);  // 0.0 not 0.0d
    }
}
extern "C" __global__ void sigmoid_f64(const double* input, double* output, int size) {
    int idx = get_global_idx();
    if (idx < size) {
        double x = input[idx];  // Should be double, not float
        output[idx] = 1.0 / (1.0 + exp(-x));  // Use exp() not expf()
    }
}

extern "C" __global__ void hyperbolic_tangent_f64(
    const double* input,
    double* output,
    int size
) {
    // Calculate thread index using standard CUDA pattern
    int idx =  get_global_idx();

    // Boundary check for thread safety
    if (idx < size) {
        double x = input[idx];

        // Use the built-in tanhf function for numerical stability
        // It handles overflow/underflow cases internally
        output[idx] = tanh(x);

        // Alternative manual implementation (commented for reference):
        // float exp_pos = expf(x);
        // float exp_neg = expf(-x);
        // output[idx] = (exp_pos - exp_neg) / (exp_pos + exp_neg);
    }
}




// =============================================================================
// SOFTMAX KERNEL: WARP REDUCTION PRIMITIVES
// =============================================================================

template <typename T>
__inline__ __device__ T warpReduceMax(T val) {
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        val = max(val, __shfl_xor_sync(0xffffffff, val, mask, 32));
    }
    return val;
}

template <typename T>
__inline__ __device__ T warpReduceSum(T val) {
#pragma unroll
    for (int mask = 16; mask > 0; mask >>= 1) {
        val += __shfl_xor_sync(0xffffffff, val, mask, 32);
    }
    return val;
}

// =============================================================================
// BLOCK REDUCTION PRIMITIVES
// =============================================================================

template <typename T>
__inline__ __device__ T blockReduceMax(T val) {
    __shared__ T shared[33];

    int lane = threadIdx.x & 0x1f;  // threadIdx.x % 32
    int wid = threadIdx.x >> 5;     // threadIdx.x / 32

    // Reduce inside the warp
    val = warpReduceMax(val);

    // Thread 0 of each warp stores on shmem
    if (lane == 0) {
        shared[wid] = val;
    }
    __syncthreads();

    // First warp makes the final reduction
    if (wid == 0) {
        val = (threadIdx.x < (blockDim.x / 32)) ? shared[lane] : -FLT_MAX;
        val = warpReduceMax(val);
    }

    return val;
}

template <typename T>
__inline__ __device__ T blockReduceSum(T val) {
    __shared__ T shared[33];

    int lane = threadIdx.x & 0x1f;
    int wid = threadIdx.x >> 5;

    val = warpReduceSum(val);

    if (lane == 0) {
        shared[wid] = val;
    }
    __syncthreads();

    if (wid == 0) {
        val = (threadIdx.x < (blockDim.x / 32)) ? shared[lane] : 0.0f;
        val = warpReduceSum(val);
    }

    return val;
}

// =============================================================================
// SOFTMAX KERNEL
// =============================================================================

extern "C" __global__ void softmax(const float* input, float* output, int N) {


    int tid = threadIdx.x;
    int global_idx = get_global_idx();

    // S1: Find global maxima using block reduction
    float local_max = -FLT_MAX;

    // Initialize local max values.
    for (int i = global_idx; i < N; i += blockDim.x * gridDim.x) {
        local_max = max(local_max, input[i]);
    }

    // Reduce the maximum
    float block_max = blockReduceMax(local_max);

    // Broadcast the maximum to all threads
    __shared__ float s_max;
    if (tid == 0) {
        s_max = block_max;
    }
    __syncthreads();

    // S2: Compute local sums
    float local_sum = 0.0f;

    // Each thread calculates its local sum value.
    for (int i = global_idx; i < N; i += blockDim.x * gridDim.x) {
        local_sum += expf(input[i] - s_max);
    }

    // Reduce sum between all blocks.
    float block_sum = blockReduceSum(local_sum);

    // Broadcast the sum to all threads.
    __shared__ float s_sum;
    if (tid == 0) {
        s_sum = block_sum;
    }
    __syncthreads();

    // S3: Compute final softmax and return the result.
    for (int i = global_idx; i < N; i += blockDim.x * gridDim.x) {
        output[i] = expf(input[i] - s_max) / s_sum;
    }
}




extern "C" __global__ void softmax_f64(const double* input, double* output, int N) {

    int tid = threadIdx.x;
    int global_idx = get_global_idx();

    // S1: Find global maxima using block reduction
    double local_max = -(double) FLT_MAX;

    // Initialize local max values.
    for (int i = global_idx; i < N; i += blockDim.x * gridDim.x) {
        local_max = max(local_max, input[i]);
    }

    // Reduce the maximum
    double block_max = blockReduceMax(local_max);

    // Broadcast the maximum to all threads
    __shared__ double s_max;
    if (tid == 0) {
        s_max = block_max;
    }
    __syncthreads();

    // S2: Compute local sums
    double local_sum = 0.0f;

    // Each thread calculates its local sum value.
    for (int i = global_idx; i < N; i += blockDim.x * gridDim.x) {
        local_sum += exp(input[i] - s_max);
    }

    // Reduce sum between all blocks.
    double block_sum = blockReduceSum(local_sum);

    // Broadcast the sum to all threads.
    __shared__ double s_sum;
    if (tid == 0) {
        s_sum = block_sum;
    }
    __syncthreads();

    // S3: Compute final softmax and return the result.
    for (int i = global_idx; i < N; i += blockDim.x * gridDim.x) {
        output[i] = exp(input[i] - s_max) / s_sum;
    }
}
