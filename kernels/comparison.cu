
#include <hip/hip_runtime.h>
// kernels/comparison.cu
// CUDA kernels for comparison operations

// Helper function to get global thread index
__device__ inline int get_global_idx() {
    return blockIdx.x * blockDim.x + threadIdx.x;
}

extern "C" __global__ void clamp(
    const float* input,
    float* output, 
    float min_val,
    float max_val,
    int size
) {
    int idx = get_global_idx();
    if (idx < size) {
        float val = input[idx];
        val = val < min_val ? min_val : val;
        val = val > max_val ? max_val : val;
        output[idx] = val;
    }
}

extern "C" __global__ void greater_equal(
    const float* a,
    const float* b,
    float* result,
    int size
) {
    int idx = get_global_idx();
    if (idx < size) {
        result[idx] = (a[idx] >= b[idx]) ? 1.0f : 0.0f;
    }
}

extern "C" __global__ void greater_equal_f64(
    const double* a,
    const double* b,
    double* result,
    int size
) {
    int idx = get_global_idx();
    if (idx < size) {
        result[idx] = (a[idx] >= b[idx]) ? 1.0 : 0.0;
    }
}

extern "C" __global__ void less_equal(
    const float* a,
    const float* b,
    float* result,
    int size
) {
    int idx = get_global_idx();
    if (idx < size) {
        result[idx] = (a[idx] <= b[idx]) ? 1.0f : 0.0f;
    }
}

extern "C" __global__ void less_equal_f64(
    const double* a,
    const double* b,
    double* result,
    int size
) {
    int idx = get_global_idx();
    if (idx < size) {
        result[idx] = (a[idx] <= b[idx]) ? 1.0 : 0.0;
    }
}

extern "C" __global__ void equal(
    const float* a,
    const float* b,
    float* result,
    int size
) {
    int idx = get_global_idx();
    if (idx < size) {
        result[idx] = (a[idx] == b[idx]) ? 1.0f : 0.0f;
    }
}

extern "C" __global__ void equal_f64(
    const double* a,
    const double* b,
    double* result,
    int size
) {
    int idx = get_global_idx();
    if (idx < size) {
        result[idx] = (a[idx] == b[idx]) ? 1.0 : 0.0;
    }
}

extern "C" __global__ void logical_not(
    const float* input,
    float* result,
    int size
) {
    int idx = get_global_idx();
    if (idx < size) {
        result[idx] = (input[idx] == 0.0f) ? 1.0f : 0.0f;
    }
}

extern "C" __global__ void logical_not_f64(
    const double* input,
    double* result,
    int size
) {
    int idx = get_global_idx();
    if (idx < size) {
        result[idx] = (input[idx] == 0.0) ? 1.0 : 0.0;
    }
}

extern "C" __global__ void in_range(
    const float* input,
    float min_val,
    float max_val,
    float* result,
    int size
) {
    int idx = get_global_idx();
    if (idx < size) {
        result[idx] = (input[idx] >= min_val && input[idx] <= max_val) ? 1.0f : 0.0f;
    }
}

extern "C" __global__ void in_range_f64(
    const double* input,
    double min_val,
    double max_val,
    double* result,
    int size
) {
    int idx = get_global_idx();
    if (idx < size) {
        result[idx] = (input[idx] >= min_val && input[idx] <= max_val) ? 1.0 : 0.0;
    }
}

// kernels/sign.cu
// CUDA kernel for sign operation

extern "C" __global__ void sign(
    const float* input,
    float* result,
    int size
) {
    int idx = get_global_idx();
    
    if (idx < size) {
        float x = input[idx];
        if (x > 0.0f) {
            result[idx] = 1.0f;
        } else if (x < 0.0f) {
            result[idx] = -1.0f;
        } else {
            result[idx] = 0.0f;  // x == 0
        }
    }
}

extern "C" __global__ void sign_kernel_f64(
    const double* input,
    double* result,
    int size
) {
    int idx = get_global_idx();
    
    if (idx < size) {
        double x = input[idx];
        if (x > 0.0) {
            result[idx] = 1.0;
        } else if (x < 0.0) {
            result[idx] = -1.0;
        } else {
            result[idx] = 0.0;  // x == 0
        }
    }
}