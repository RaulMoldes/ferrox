
#include <hip/hip_runtime.h>
extern "C" __global__ void sigmoid_forward(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        output[idx] = 1.0f / (1.0f + expf(-x));
    }
}
